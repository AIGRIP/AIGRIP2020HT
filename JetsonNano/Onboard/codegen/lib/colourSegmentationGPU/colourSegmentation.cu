#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: colourSegmentation.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 04-Nov-2020 11:21:24
//

// Include Files
#include "colourSegmentation.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWPtxUtils.hpp"
#include "gpu_imreconstruct_types.hpp"
#include "imreconstruct_cuda.hpp"
#include "smemutil.hpp"
#include <cmath>

// Type Definitions
struct cell_wrap_7
{
  double f1[9];
};

// Function Declarations
static __global__ void colourSegmentation_kernel1(const unsigned char
  colourBalancedImage[2184480], double Xin[2184480]);
static __global__ void colourSegmentation_kernel10(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel11(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel12(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel13(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel14(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel15(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel16(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel17(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel18(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel19(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel2(const double Xin[2184480],
  double HSVImage[2184480]);
static __global__ void colourSegmentation_kernel20(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel21(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel22(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel23(const bool centerColourMask
  [23409], cell_wrap_7 outputs[1]);
static __global__ void colourSegmentation_kernel24(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel25(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel26(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel27(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel28(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel29(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel3(double HSVImage[2184480]);
static __global__ void colourSegmentation_kernel30(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel31(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel32(const double HSVImage[2184480],
  bool colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel33(const bool
  colourSegmentationMask[728160], signed char mask[728160]);
static __global__ void colourSegmentation_kernel34(unsigned char maskPad[731612]);
static __global__ void colourSegmentation_kernel35(unsigned char maskPad[731612]);
static __global__ void colourSegmentation_kernel36(const signed char mask[728160],
  unsigned char maskPad[731612]);
static __global__ void colourSegmentation_kernel37(unsigned char markerPad
  [731612]);
static __global__ void colourSegmentation_kernel38(unsigned char markerPad
  [731612]);
static __global__ void colourSegmentation_kernel39(unsigned char markerPad
  [731612]);
static __global__ void colourSegmentation_kernel4(bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel40(short idx[1968]);
static __global__ void colourSegmentation_kernel41(short idx[1968]);
static __global__ void colourSegmentation_kernel42(unsigned char markerPad
  [731612], unsigned char maskPad[731612]);
static __global__ void colourSegmentation_kernel43(unsigned char I1[731612]);
static __global__ void colourSegmentation_kernel44(const short idx[1968], bool
  x[984]);
static __global__ void colourSegmentation_kernel45(const bool x[984], int *nz);
static __global__ void colourSegmentation_kernel46(const bool x[984], int *nz);
static __global__ void colourSegmentation_kernel47(const short idx[1968], bool
  x[984]);
static __global__ void colourSegmentation_kernel48(const bool x[984], int *nz);
static __global__ void colourSegmentation_kernel49(const bool x[984], int *nz);
static __global__ void colourSegmentation_kernel5(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel50(const unsigned char I1[731612],
  const short idx[1968], const int *nz, const int *b_nz, bool
  colourSegmentationMask[728160]);
static __global__ void colourSegmentation_kernel51(const double ex, double
  r_data[1]);
static __global__ void colourSegmentation_kernel52(const double x, double
  c_data[1]);
static __global__ void colourSegmentation_kernel53(const double x, const double
  ex, bool b_data[1]);
static __global__ void colourSegmentation_kernel54(bool b_data[1]);
static __global__ void colourSegmentation_kernel55(int *nz);
static __global__ void colourSegmentation_kernel56(bool b_data[1]);
static __global__ void colourSegmentation_kernel57(int *nz);
static __global__ void colourSegmentation_kernel58(const double c_data[1], const
  double r_data[1], const int r_size[2], int seed_indices_data[1]);
static __global__ void colourSegmentation_kernel59(const bool
  colourSegmentationMask[728160], bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel6(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel60(const int seed_indices_data[1],
  int locationsVar_data[1]);
static __global__ void colourSegmentation_kernel61(const int seed_indices_data[1],
  const int seed_indices_size[1], bool b_data[1]);
static __global__ void colourSegmentation_kernel62(const bool SegmentationMask
  [728160], bool marker[728160], bool mask[728160]);
static __global__ void colourSegmentation_kernel63(const bool mask[728160],
  const int locationsVar_data[1], const int locationsVar_size[1], bool marker
  [728160]);
static __global__ void colourSegmentation_kernel64(const bool marker[728160],
  bool marker_tmp[728160]);
static __global__ void colourSegmentation_kernel65(const bool
  colourSegmentationMask[728160], const bool marker[728160], bool
  SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel7(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel8(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel9(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static double rt_roundd_snf(double u);
static __device__ int shflDown1(int in1, unsigned int offset, unsigned int mask);
static __device__ int threadGroupReduction(int val, unsigned int lane, unsigned
  int mask);
static __device__ int workGroupReduction(int val, unsigned int mask, unsigned
  int numActiveWarps);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char colourBalancedImage[2184480]
//                double Xin[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel1(
  const unsigned char colourBalancedImage[2184480], double Xin[2184480])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2184480) {
    // Input:
    // colourBalancedImage = RGB image 984x740x3
    // centerOfObjectX = Objects center postion in the x-axis
    // centerOfObjectY = Objects center postion in the y-axis
    // Output:
    // SegmentationMask = bitmask of the segmented object
    // Transform the Image into the HSV colour space
    Xin[i] = static_cast<double>(colourBalancedImage[i]) / 255.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel10(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 5202] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel11(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] > 70.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + c_i)] < 155.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] == 70.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel12(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 7803] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel13(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] > 155.0)) && (static_cast<int>(HSVImage
      [(i + xpageoffset) + 984 * (i2 + c_i)] < 205.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] == 155.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel14(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 10404] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel15(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] > 205.0)) && (static_cast<int>(HSVImage
      [(i + xpageoffset) + 984 * (i2 + c_i)] < 260.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] == 205.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel16(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 13005] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel17(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] > 260.0)) && (static_cast<int>(HSVImage
      [(i + xpageoffset) + 984 * (i2 + c_i)] < 315.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] == 260.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel18(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 15606] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel19(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 1456320]
                       > 0.6)) || (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] == 0.6)))) && (static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       < 0.2)) || (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 728160] == 0.2)))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Xin[2184480]
//                double HSVImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel2(
  const double Xin[2184480], double HSVImage[2184480])
{
  double varargin_1[3];
  double h;
  double tmp;
  unsigned long threadId;
  int k;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  k = static_cast<int>(threadId % 984UL);
  xpageoffset = static_cast<int>((threadId - static_cast<unsigned long>(k)) /
    984UL);
  if ((static_cast<int>(xpageoffset < 740)) && (static_cast<int>(k < 984))) {
    double d;
    double d1;
    double d2;
    double delta;
    double v;
    int i;
    d = Xin[k + 984 * xpageoffset];
    varargin_1[0] = d;
    d1 = Xin[(k + 984 * xpageoffset) + 728160];
    varargin_1[1] = d1;
    d2 = Xin[(k + 984 * xpageoffset) + 1456320];
    varargin_1[2] = d2;
    v = d;
    for (i = 0; i < 2; i++) {
      if (v < varargin_1[i + 1]) {
        v = varargin_1[i + 1];
      }
    }

    varargin_1[0] = d;
    varargin_1[1] = d1;
    varargin_1[2] = d2;
    tmp = d;
    for (i = 0; i < 2; i++) {
      if (tmp > varargin_1[i + 1]) {
        tmp = varargin_1[i + 1];
      }
    }

    delta = v - tmp;
    tmp = delta;
    if (delta == 0.0) {
      tmp = 1.0;
    }

    h = 0.0;
    if (d == v) {
      h = (d1 - d2) / tmp;
    }

    if (d1 == v) {
      h = (d2 - d) / tmp + 2.0;
    }

    if (d2 == v) {
      h = (d - d1) / tmp + 4.0;
    }

    h /= 6.0;
    if (h < 0.0) {
      h++;
    }

    tmp /= v;
    if (delta == 0.0) {
      h = 0.0;
      tmp = 0.0;
    }

    if (!static_cast<int>(v != 0.0)) {
      tmp = 0.0;
    }

    HSVImage[k + 984 * xpageoffset] = h;
    HSVImage[(k + 984 * xpageoffset) + 728160] = tmp;
    HSVImage[(k + 984 * xpageoffset) + 1456320] = v;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel20(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 18207] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel21(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 1456320]
                       < 0.3)) || (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] == 0.3)))) || (static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 1456320]
                       < 0.6)) && (static_cast<int>((static_cast<int>(HSVImage
      [((i + xpageoffset) + 984 * (i2 + c_i)) + 728160] < 0.2)) || (static_cast<
      int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160] == 0.2)))))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel22(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 20808] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool centerColourMask[23409]
//                cell_wrap_7 outputs[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel23(
  const bool centerColourMask[23409], cell_wrap_7 outputs[1])
{
  double d;
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 9) {
    int xpageoffset;

    //  Sum up all the masks and then determine which mask has the most pixels
    xpageoffset = i * 2601;
    d = static_cast<double>(centerColourMask[xpageoffset]);
    for (int k = 0; k < 2600; k++) {
      d += static_cast<double>(centerColourMask[(xpageoffset + k) + 1]);
    }

    outputs[0].f1[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel24(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    double d1;
    d = HSVImage[(xpageoffset + 984 * i) + 1456320];
    d1 = HSVImage[(xpageoffset + 984 * i) + 728160];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>(d < 0.3)) || (static_cast<int>(d == 0.3)))) || (
      static_cast<int>((static_cast<int>(d < 0.6)) && (static_cast<int>((
      static_cast<int>(d1 < 0.2)) || (static_cast<int>(d1 == 0.2)))))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel25(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    double d1;
    d = HSVImage[(xpageoffset + 984 * i) + 1456320];
    d1 = HSVImage[(xpageoffset + 984 * i) + 728160];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>(d > 0.6)) || (static_cast<int>(d == 0.6)))) && (
      static_cast<int>((static_cast<int>(d1 < 0.2)) || (static_cast<int>(d1 ==
      0.2)))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel26(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d > 260.0)) && (
      static_cast<int>(d < 315.0)))) || (static_cast<int>(d == 260.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel27(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d > 205.0)) && (
      static_cast<int>(d < 260.0)))) || (static_cast<int>(d == 205.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel28(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d > 155.0)) && (
      static_cast<int>(d < 205.0)))) || (static_cast<int>(d == 155.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel29(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d > 70.0)) && (
      static_cast<int>(d < 155.0)))) || (static_cast<int>(d == 70.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double HSVImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel3
  (double HSVImage[2184480])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    HSVImage[xpageoffset + 984 * i] *= 360.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel30(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d > 50.0)) && (
      static_cast<int>(d < 70.0)))) || (static_cast<int>(d == 50.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel31(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d > 15.0)) && (
      static_cast<int>(d < 50.0)))) || (static_cast<int>(d == 15.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel32(
  const double HSVImage[2184480], bool colourSegmentationMask[728160])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       984UL);
  if ((static_cast<int>(i < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * i];
    colourSegmentationMask[xpageoffset + 984 * i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(d < 15.0)) || (
      static_cast<int>(d == 315.0)))) || (static_cast<int>(d > 315.0)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 728160] > 0.2)))) && (
      static_cast<int>(HSVImage[(xpageoffset + 984 * i) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool colourSegmentationMask[728160]
//                signed char mask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel33(
  const bool colourSegmentationMask[728160], signed char mask[728160])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 728160) {
    // Fill up the hole in the binary image.
    mask[i] = static_cast<signed char>(colourSegmentationMask[i]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char maskPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel34
  (unsigned char maskPad[731612])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 986) {
    maskPad[i] = static_cast<unsigned char>(0U);
    maskPad[i + 730626] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char maskPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel35
  (unsigned char maskPad[731612])
{
  int xpageoffset;
  xpageoffset = static_cast<int>(mwGetGlobalThreadIndex());
  if (xpageoffset < 740) {
    maskPad[986 * (xpageoffset + 1)] = static_cast<unsigned char>(0U);
    maskPad[986 * (xpageoffset + 1) + 985] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char mask[728160]
//                unsigned char maskPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel36(
  const signed char mask[728160], unsigned char maskPad[731612])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  i = static_cast<int>(threadId % 984UL);
  xpageoffset = static_cast<int>((threadId - static_cast<unsigned long>(i)) /
    984UL);
  if ((static_cast<int>(xpageoffset < 740)) && (static_cast<int>(i < 984))) {
    maskPad[(i + 986 * (xpageoffset + 1)) + 1] = static_cast<unsigned char>
      (mask[i + 984 * xpageoffset]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char markerPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel37
  (unsigned char markerPad[731612])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 986) {
    markerPad[i] = static_cast<unsigned char>(0U);
    markerPad[i + 730626] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char markerPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel38
  (unsigned char markerPad[731612])
{
  int xpageoffset;
  xpageoffset = static_cast<int>(mwGetGlobalThreadIndex());
  if (xpageoffset < 740) {
    markerPad[986 * (xpageoffset + 1)] = static_cast<unsigned char>(0U);
    markerPad[986 * (xpageoffset + 1) + 985] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char markerPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel39
  (unsigned char markerPad[731612])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  i = static_cast<int>(threadId % 984UL);
  xpageoffset = static_cast<int>((threadId - static_cast<unsigned long>(i)) /
    984UL);
  if ((static_cast<int>(xpageoffset < 740)) && (static_cast<int>(i < 984))) {
    markerPad[(i + 986 * (xpageoffset + 1)) + 1] = MAX_uint8_T;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel4(bool
  centerColourMask[23409])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 23409) {
    centerColourMask[i] = false;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                short idx[1968]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel40
  (short idx[1968])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 1968) {
    idx[i] = static_cast<short>(0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                short idx[1968]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel41
  (short idx[1968])
{
  int k;
  short iv[1];
  k = static_cast<int>(mwGetGlobalThreadIndex());
  if (k < 2) {
    iv[0] = static_cast<short>(-244 * k + 984);
    for (int i = 0; i < static_cast<int>(iv[0]); i++) {
      idx[i + 984 * k] = static_cast<short>(i + 2);
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char markerPad[731612]
//                unsigned char maskPad[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel42
  (unsigned char markerPad[731612], unsigned char maskPad[731612])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 731612) {
    maskPad[i] = static_cast<unsigned char>(255U - static_cast<unsigned int>(
      static_cast<int>(maskPad[i])));
    markerPad[i] = static_cast<unsigned char>(255U - static_cast<unsigned int>(
      static_cast<int>(markerPad[i])));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned char I1[731612]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel43
  (unsigned char I1[731612])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 731612) {
    I1[i] = static_cast<unsigned char>(255U - static_cast<unsigned int>(
      static_cast<int>(I1[i])));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short idx[1968]
//                bool x[984]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel44(
  const short idx[1968], bool x[984])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 984) {
    x[i] = (static_cast<int>(idx[i]) != 0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool x[984]
//                int *nz
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel45(
  const bool x[984], int *nz)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *nz = static_cast<int>(x[0]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool x[984]
//                int *nz
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel46(
  const bool x[984], int *nz)
{
  unsigned int blockStride;
  unsigned int idx;
  unsigned int mask;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  int tmpRed0;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0;
  if (mwIsLastBlock()) {
    int m;
    m = static_cast<int>(983U % blockStride);
    if (static_cast<unsigned int>(m) > 0U) {
      blockStride = static_cast<unsigned int>(m);
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 982U) {
    tmpRed0 = static_cast<int>(x[static_cast<int>(threadId) + 1]);
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 982U);
  for (idx = threadId + threadStride; idx <= 982U; idx += threadStride) {
    tmpRed0 += static_cast<int>(x[static_cast<int>(idx) + 1]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, blockStride);
  if (thBlkId == 0U) {
    atomicAdd(&nz[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short idx[1968]
//                bool x[984]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel47(
  const short idx[1968], bool x[984])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 984) {
    x[i] = (static_cast<int>(idx[i + 984]) != 0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool x[984]
//                int *nz
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel48(
  const bool x[984], int *nz)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *nz = static_cast<int>(x[0]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool x[984]
//                int *nz
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel49(
  const bool x[984], int *nz)
{
  unsigned int blockStride;
  unsigned int idx;
  unsigned int mask;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  int tmpRed0;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0;
  if (mwIsLastBlock()) {
    int m;
    m = static_cast<int>(983U % blockStride);
    if (static_cast<unsigned int>(m) > 0U) {
      blockStride = static_cast<unsigned int>(m);
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 982U) {
    tmpRed0 = static_cast<int>(x[static_cast<int>(threadId) + 1]);
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 982U);
  for (idx = threadId + threadStride; idx <= 982U; idx += threadStride) {
    tmpRed0 += static_cast<int>(x[static_cast<int>(idx) + 1]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, blockStride);
  if (thBlkId == 0U) {
    atomicAdd(&nz[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel5(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] < 15.0)) || (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + c_i)] == 315.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] > 315.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char I1[731612]
//                const short idx[1968]
//                const int *nz
//                const int *b_nz
//                bool colourSegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel50(
  const unsigned char I1[731612], const short idx[1968], const int *nz, const
  int *b_nz, bool colourSegmentationMask[728160])
{
  unsigned long b_idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(*nz - 1) + 1L) * (static_cast<long>(*b_nz - 1) +
    1L) - 1L;
  for (b_idx = threadId; b_idx <= static_cast<unsigned long>(loopEnd); b_idx +=
       threadStride) {
    int i;
    int xpageoffset;
    xpageoffset = static_cast<int>(b_idx % (static_cast<unsigned long>(*nz - 1)
      + 1UL));
    i = static_cast<int>((b_idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(*nz - 1) + 1UL));
    colourSegmentationMask[i + 984 * xpageoffset] = (static_cast<int>(I1[(
      static_cast<int>(idx[i]) + 986 * (static_cast<int>(idx[xpageoffset + 984])
      - 1)) - 1]) != 0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ex
//                double r_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel51(
  const double ex, double r_data[1])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    r_data[0] = ex;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double x
//                double c_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel52(
  const double x, double c_data[1])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c_data[0] = x;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double x
//                const double ex
//                bool b_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel53(
  const double x, const double ex, bool b_data[1])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0] = ((static_cast<int>((static_cast<int>(ex < 1.0)) || (static_cast<
      int>(ex > 984.0)))) || (static_cast<int>(x < 1.0)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                bool b_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel54(bool
  b_data[1])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0] = true;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                int *nz
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel55(int *
  nz)
{
  unsigned int blockStride;
  unsigned int idx;
  unsigned int mask;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  int tmpRed0;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0;
  if (mwIsLastBlock()) {
    int m;
    m = static_cast<int>(1U % blockStride);
    if (static_cast<unsigned int>(m) > 0U) {
      blockStride = 1U;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 0U) {
    tmpRed0 = 1;
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 0U);
  for (idx = threadId + threadStride; idx <= 0U; idx += threadStride) {
    tmpRed0++;
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, blockStride);
  if (thBlkId == 0U) {
    atomicAdd(&nz[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                bool b_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel56(bool
  b_data[1])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0] = true;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                int *nz
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel57(int *
  nz)
{
  unsigned int blockStride;
  unsigned int idx;
  unsigned int mask;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  int tmpRed0;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0;
  if (mwIsLastBlock()) {
    int m;
    m = static_cast<int>(1U % blockStride);
    if (static_cast<unsigned int>(m) > 0U) {
      blockStride = 1U;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 0U) {
    tmpRed0 = 1;
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 0U);
  for (idx = threadId + threadStride; idx <= 0U; idx += threadStride) {
    tmpRed0++;
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, blockStride);
  if (thBlkId == 0U) {
    atomicAdd(&nz[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double c_data[1]
//                const double r_data[1]
//                const int r_size[2]
//                int seed_indices_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel58(
  const double c_data[1], const double r_data[1], const int r_size[2], int
  seed_indices_data[1])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<long>(r_size[1] - 1);
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    seed_indices_data[0] = static_cast<int>(r_data[0]) + 984 * (static_cast<int>
      (c_data[0]) - 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool colourSegmentationMask[728160]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel59(
  const bool colourSegmentationMask[728160], bool SegmentationMask[728160])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 728160) {
    SegmentationMask[i] = static_cast<bool>(!static_cast<int>
      (colourSegmentationMask[i]));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel6(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[xpageoffset + 51 * i] = HSVImage_data[xpageoffset + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int seed_indices_data[1]
//                int locationsVar_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel60(
  const int seed_indices_data[1], int locationsVar_data[1])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 1) {
    locationsVar_data[0] = seed_indices_data[0];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int seed_indices_data[1]
//                const int seed_indices_size[1]
//                bool b_data[1]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel61(
  const int seed_indices_data[1], const int seed_indices_size[1], bool b_data[1])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<long>(seed_indices_size[0] - 1);
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    b_data[0] = ((static_cast<int>(seed_indices_data[0] < 1)) || (static_cast<
      int>(seed_indices_data[0] > 728160)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool SegmentationMask[728160]
//                bool marker[728160]
//                bool mask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel62(
  const bool SegmentationMask[728160], bool marker[728160], bool mask[728160])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 728160) {
    mask[i] = static_cast<bool>(!static_cast<int>(SegmentationMask[i]));
    marker[i] = false;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool mask[728160]
//                const int locationsVar_data[1]
//                const int locationsVar_size[1]
//                bool marker[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel63(
  const bool mask[728160], const int locationsVar_data[1], const int
  locationsVar_size[1], bool marker[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<long>(locationsVar_size[0] - 1);
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    marker[locationsVar_data[0] - 1] = mask[locationsVar_data[0] - 1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool marker[728160]
//                bool marker_tmp[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel64(
  const bool marker[728160], bool marker_tmp[728160])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 728160) {
    marker_tmp[i] = marker[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool colourSegmentationMask[728160]
//                const bool marker[728160]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel65(
  const bool colourSegmentationMask[728160], const bool marker[728160], bool
  SegmentationMask[728160])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 728160) {
    SegmentationMask[i] = ((static_cast<int>((static_cast<int>
      (SegmentationMask[i])) || (static_cast<int>(marker[i])))) && (static_cast<
      int>(colourSegmentationMask[i])));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel7(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] > 15.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + c_i)] < 50.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] == 15.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel8(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int i;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  i = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                       51UL);
  if ((static_cast<int>(i < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * i) + 2601] = HSVImage_data[xpageoffset
      + 51 * i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel9(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int c_i;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    c_i = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * c_i] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + c_i)] > 50.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + c_i)] < 70.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + c_i)] == 50.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + c_i)) + 728160]
                       > 0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + c_i)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : double u
// Return Type  : double
//
static double rt_roundd_snf(double u)
{
  double y;
  if (std::abs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = std::floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = std::ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

//
// Arguments    : int in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : int
//
static __device__ int shflDown1(int in1, unsigned int offset, unsigned int mask)
{
  in1 = __shfl_down_sync(mask, in1, offset);
  return in1;
}

//
// Arguments    : int val
//                unsigned int lane
//                unsigned int mask
// Return Type  : int
//
static __device__ int threadGroupReduction(int val, unsigned int lane, unsigned
  int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    int other;
    other = shflDown1(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }

    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }

  return val;
}

//
// Arguments    : int val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : int
//
static __device__ int workGroupReduction(int val, unsigned int mask, unsigned
  int numActiveWarps)
{
  __shared__ int shared[32];
  unsigned int lane;
  unsigned int thBlkId;
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }

  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }

  return val;
}

//
// The function takes in a image and a rough postion of the object in the
// image that should be segmented. and it outputs a a bitmask of the the
// object.
// Arguments    : const unsigned char colourBalancedImage[2184480]
//                double centerOfObjectX
//                double centerOfObjectY
//                bool SegmentationMask[728160]
// Return Type  : void
//
void colourSegmentation(const unsigned char colourBalancedImage[2184480], double
  centerOfObjectX, double centerOfObjectY, bool SegmentationMask[728160])
{
  cell_wrap_7 outputs[1];
  cell_wrap_7 (*gpu_outputs)[1];
  dim3 b_block;
  dim3 b_grid;
  dim3 block;
  dim3 c_block;
  dim3 c_grid;
  dim3 d_block;
  dim3 d_grid;
  dim3 e_block;
  dim3 e_grid;
  dim3 f_block;
  dim3 f_grid;
  dim3 g_block;
  dim3 g_grid;
  dim3 grid;
  dim3 h_block;
  dim3 h_grid;
  dim3 i_block;
  dim3 i_grid;
  dim3 j_block;
  dim3 j_grid;
  dim3 k_block;
  dim3 k_grid;
  dim3 l_block;
  dim3 l_grid;
  dim3 m_block;
  dim3 m_grid;
  double (*gpu_HSVImage)[2184480];
  double (*gpu_Xin)[2184480];
  double (*gpu_c_data)[1];
  double (*gpu_r_data)[1];
  double ex;
  double x;
  int HSVImage_size[2];
  int b_HSVImage_size[2];
  int b_size[2];
  int c_HSVImage_size[2];
  int d_HSVImage_size[2];
  int e_HSVImage_size[2];
  int f_HSVImage_size[2];
  int g_HSVImage_size[2];
  int h_HSVImage_size[2];
  int i_HSVImage_size[2];
  int r_size[2];
  int (*b_gpu_HSVImage_size)[2];
  int (*c_gpu_HSVImage_size)[2];
  int (*d_gpu_HSVImage_size)[2];
  int (*e_gpu_HSVImage_size)[2];
  int (*f_gpu_HSVImage_size)[2];
  int (*g_gpu_HSVImage_size)[2];
  int (*gpu_HSVImage_size)[2];
  int (*gpu_b_size)[2];
  int (*gpu_r_size)[2];
  int (*h_gpu_HSVImage_size)[2];
  int (*i_gpu_HSVImage_size)[2];
  int locationsVar_data[1];
  int locationsVar_size[1];
  int seed_indices_size[1];
  int (*gpu_locationsVar_data)[1];
  int (*gpu_locationsVar_size)[1];
  int (*gpu_seed_indices_data)[1];
  int (*gpu_seed_indices_size)[1];
  int b_i;
  int b_nz;
  int i;
  int i1;
  int i2;
  int nz;
  int *b_gpu_nz;
  int *gpu_nz;
  short (*gpu_idx)[1968];
  unsigned char (*gpu_colourBalancedImage)[2184480];
  unsigned char (*gpu_I1)[731612];
  unsigned char (*gpu_markerPad)[731612];
  unsigned char (*gpu_maskPad)[731612];
  signed char (*b_gpu_mask)[728160];
  signed char badPixels_size[1];
  bool (*gpu_HSVImage_data)[728160];
  bool (*gpu_SegmentationMask)[728160];
  bool (*gpu_colourSegmentationMask)[728160];
  bool (*gpu_marker)[728160];
  bool (*gpu_marker_tmp)[728160];
  bool (*gpu_mask)[728160];
  bool (*gpu_centerColourMask)[23409];
  bool (*gpu_x)[984];
  bool b_data[1];
  bool (*gpu_b_data)[1];
  bool b_data_dirtyOnGpu;
  bool exitg1;
  bool locationsVar_data_dirtyOnCpu;
  bool locationsVar_data_dirtyOnGpu;
  bool validLaunchParams;
  hipMalloc(&gpu_marker_tmp, 728160UL);
  hipMalloc(&gpu_marker, 728160UL);
  hipMalloc(&gpu_mask, 728160UL);
  hipMalloc(&gpu_SegmentationMask, 728160UL);
  hipMalloc(&gpu_locationsVar_data, 4UL);
  hipMalloc(&gpu_seed_indices_data, 4UL);
  hipMalloc(&gpu_locationsVar_size, 4UL);
  hipMalloc(&gpu_seed_indices_size, 4UL);
  hipMalloc(&gpu_c_data, 8UL);
  hipMalloc(&gpu_r_data, 8UL);
  hipMalloc(&gpu_b_data, 1UL);
  hipMalloc(&gpu_b_size, 8UL);
  hipMalloc(&gpu_r_size, 8UL);
  hipMalloc(&gpu_x, 984UL);
  hipMalloc(&gpu_I1, 731612UL);
  hipMalloc(&gpu_idx, 3936UL);
  hipMalloc(&gpu_markerPad, 731612UL);
  hipMalloc(&b_gpu_mask, 728160UL);
  hipMalloc(&gpu_maskPad, 731612UL);
  hipMalloc(&gpu_colourSegmentationMask, 728160UL);
  hipMalloc(&gpu_outputs, 72UL);
  hipMalloc(&i_gpu_HSVImage_size, 8UL);
  hipMalloc(&h_gpu_HSVImage_size, 8UL);
  hipMalloc(&g_gpu_HSVImage_size, 8UL);
  hipMalloc(&f_gpu_HSVImage_size, 8UL);
  hipMalloc(&e_gpu_HSVImage_size, 8UL);
  hipMalloc(&d_gpu_HSVImage_size, 8UL);
  hipMalloc(&c_gpu_HSVImage_size, 8UL);
  hipMalloc(&b_gpu_HSVImage_size, 8UL);
  hipMalloc(&gpu_HSVImage_data, 728160UL);
  hipMalloc(&gpu_centerColourMask, 23409UL);
  hipMalloc(&gpu_HSVImage_size, 8UL);
  hipMalloc(&gpu_nz, 4UL);
  hipMalloc(&b_gpu_nz, 4UL);
  hipMalloc(&gpu_HSVImage, 17475840UL);
  hipMalloc(&gpu_Xin, 17475840UL);
  hipMalloc(&gpu_colourBalancedImage, 2184480UL);
  locationsVar_data_dirtyOnCpu = false;
  locationsVar_data_dirtyOnGpu = false;

  // Input:
  // colourBalancedImage = RGB image 984x740x3
  // centerOfObjectX = Objects center postion in the x-axis
  // centerOfObjectY = Objects center postion in the y-axis
  // Output:
  // SegmentationMask = bitmask of the segmented object
  // Transform the Image into the HSV colour space
  hipMemcpy(gpu_colourBalancedImage, (void *)&colourBalancedImage[0], 2184480UL,
             hipMemcpyHostToDevice);
  colourSegmentation_kernel1<<<dim3(4267U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_colourBalancedImage, *gpu_Xin);
  colourSegmentation_kernel2<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_Xin, *gpu_HSVImage);

  // Standardize the H-values from 0-1 to 0-360
  colourSegmentation_kernel3<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage);

  // Make a small square at the center of the object and determine which
  // colour is most prevalent in the square
  // Choose square size
  // Calculate the start and end values for X and  Y
  // Take out the sqaure area from the image
  if (centerOfObjectX - 25.0 > centerOfObjectX + 25.0) {
    i = 0;
    nz = 0;
  } else {
    i = static_cast<int>(centerOfObjectX - 25.0) - 1;
    nz = static_cast<int>(centerOfObjectX + 25.0);
  }

  if (centerOfObjectY - 25.0 > centerOfObjectY + 25.0) {
    i2 = 0;
    b_nz = 0;
  } else {
    i2 = static_cast<int>(centerOfObjectY - 25.0) - 1;
    b_nz = static_cast<int>(centerOfObjectY + 25.0);
  }

  colourSegmentation_kernel4<<<dim3(46U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_centerColourMask);

  // Make mask in the square for every 9 colours.
  // Calcualte which pixels are red
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  HSVImage_size[0] = b_i + 1;
  HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_HSVImage_size, &HSVImage_size[0], 8UL, hipMemcpyHostToDevice);
    colourSegmentation_kernel5<<<grid, block>>>(*gpu_HSVImage, i2, i,
      *gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel6<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are orange
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  b_HSVImage_size[0] = b_i + 1;
  b_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &b_grid, &b_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(b_gpu_HSVImage_size, &b_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel7<<<b_grid, b_block>>>(*gpu_HSVImage, i2, i,
      *b_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel8<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are yellow
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  c_HSVImage_size[0] = b_i + 1;
  c_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &c_grid, &c_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(c_gpu_HSVImage_size, &c_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel9<<<c_grid, c_block>>>(*gpu_HSVImage, i2, i,
      *c_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel10<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are green
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  d_HSVImage_size[0] = b_i + 1;
  d_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &d_grid, &d_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(d_gpu_HSVImage_size, &d_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel11<<<d_grid, d_block>>>(*gpu_HSVImage, i2, i,
      *d_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel12<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are cyan
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  e_HSVImage_size[0] = b_i + 1;
  e_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &e_grid, &e_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(e_gpu_HSVImage_size, &e_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel13<<<e_grid, e_block>>>(*gpu_HSVImage, i2, i,
      *e_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel14<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are blue
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  f_HSVImage_size[0] = b_i + 1;
  f_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &f_grid, &f_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(f_gpu_HSVImage_size, &f_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel15<<<f_grid, f_block>>>(*gpu_HSVImage, i2, i,
      *f_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel16<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are purple
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  g_HSVImage_size[0] = b_i + 1;
  g_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &g_grid, &g_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(g_gpu_HSVImage_size, &g_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel17<<<g_grid, g_block>>>(*gpu_HSVImage, i2, i,
      *g_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel18<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are white
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  h_HSVImage_size[0] = b_i + 1;
  h_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &h_grid, &h_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(h_gpu_HSVImage_size, &h_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel19<<<h_grid, h_block>>>(*gpu_HSVImage, i2, i,
      *h_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel20<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are black
  b_i = (nz - i) - 1;
  i1 = (b_nz - i2) - 1;
  i_HSVImage_size[0] = b_i + 1;
  i_HSVImage_size[1] = i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) * (i1
    + 1L)), &i_grid, &i_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(i_gpu_HSVImage_size, &i_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel21<<<i_grid, i_block>>>(*gpu_HSVImage, i2, i,
      *i_gpu_HSVImage_size, b_i, i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel22<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  //  Sum up all the masks and then determine which mask has the most pixels
  colourSegmentation_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_centerColourMask, *gpu_outputs);
  nz = 1;
  hipMemcpy(&outputs[0], gpu_outputs, 72UL, hipMemcpyDeviceToHost);
  validLaunchParams = false;
  ex = outputs[0].f1[0];
  for (i = 0; i < 8; i++) {
    if (validLaunchParams) {
      hipMemcpy(&outputs[0], gpu_outputs, 72UL, hipMemcpyDeviceToHost);
      validLaunchParams = false;
    }

    if (ex < outputs[0].f1[i + 1]) {
      ex = outputs[0].f1[i + 1];
      nz = i + 2;
    }
  }

  //  Make a mask over the whole image. The colour that is seperated
  //  is the one that had the most pixels in the square
  if (nz == 1) {
    // Red
    colourSegmentation_kernel32<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 2) {
    // Orange
    colourSegmentation_kernel31<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 3) {
    // Yellow
    colourSegmentation_kernel30<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 4) {
    // Green
    colourSegmentation_kernel29<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 5) {
    // Cyan
    colourSegmentation_kernel28<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 6) {
    // Blue
    colourSegmentation_kernel27<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 7) {
    // Purple
    colourSegmentation_kernel26<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else if (nz == 8) {
    // White
    colourSegmentation_kernel25<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  } else {
    // Black
    colourSegmentation_kernel24<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_colourSegmentationMask);
  }

  // Fill up the hole in the binary image.
  colourSegmentation_kernel33<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_colourSegmentationMask, *b_gpu_mask);
  colourSegmentation_kernel34<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_maskPad);
  colourSegmentation_kernel35<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_maskPad);
  colourSegmentation_kernel36<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*b_gpu_mask, *gpu_maskPad);
  colourSegmentation_kernel37<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_markerPad);
  colourSegmentation_kernel38<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_markerPad);
  colourSegmentation_kernel39<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_markerPad);
  colourSegmentation_kernel40<<<dim3(4U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_idx);
  colourSegmentation_kernel41<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_idx);
  colourSegmentation_kernel42<<<dim3(1429U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_markerPad, *gpu_maskPad);
  imreconstruct_cuda(&(*gpu_markerPad)[0], &(*gpu_maskPad)[0], 986U, 742U, 4UL,
                     &(*gpu_I1)[0]);
  colourSegmentation_kernel43<<<dim3(1429U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_I1);
  colourSegmentation_kernel44<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_idx,
    *gpu_x);
  hipMemcpy(gpu_nz, &b_nz, 4UL, hipMemcpyHostToDevice);
  colourSegmentation_kernel45<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x,
    gpu_nz);
  colourSegmentation_kernel46<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_x,
    gpu_nz);
  colourSegmentation_kernel47<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_idx,
    *gpu_x);
  hipMemcpy(b_gpu_nz, &nz, 4UL, hipMemcpyHostToDevice);
  colourSegmentation_kernel48<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x,
    b_gpu_nz);
  colourSegmentation_kernel49<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_x,
    b_gpu_nz);
  hipMemcpy(&b_nz, gpu_nz, 4UL, hipMemcpyDeviceToHost);
  hipMemcpy(&nz, b_gpu_nz, 4UL, hipMemcpyDeviceToHost);
  validLaunchParams = mwGetLaunchParameters(static_cast<double>(((nz - 1) + 1L) *
    ((b_nz - 1) + 1L)), &j_grid, &j_block, 1024U, 65535U);
  if (validLaunchParams) {
    colourSegmentation_kernel50<<<j_grid, j_block>>>(*gpu_I1, *gpu_idx, b_gpu_nz,
      gpu_nz, *gpu_colourSegmentationMask);
  }

  ex = rt_roundd_snf(centerOfObjectX);
  r_size[0] = 1;
  r_size[1] = 1;
  colourSegmentation_kernel51<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(ex,
    *gpu_r_data);
  x = rt_roundd_snf(centerOfObjectY);
  colourSegmentation_kernel52<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(x,
    *gpu_c_data);
  colourSegmentation_kernel53<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(x, ex,
    *gpu_b_data);
  nz = 0;
  hipMemcpy(&b_data[0], gpu_b_data, 1UL, hipMemcpyDeviceToHost);
  b_data_dirtyOnGpu = false;
  if (b_data[0] || (x > 740.0)) {
    nz = 1;
  }

  if (nz != 0) {
    b_size[0] = 1;
    b_size[1] = 1;
    colourSegmentation_kernel54<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_b_data);
    nz = 0;
    hipMemcpy(b_gpu_nz, &nz, 4UL, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b_size, &b_size[0], 8UL, hipMemcpyHostToDevice);
    colourSegmentation_kernel55<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (b_gpu_nz);
    hipMemcpy(&nz, b_gpu_nz, 4UL, hipMemcpyDeviceToHost);
    r_size[1] = (1 <= 1 - nz);
    b_size[0] = 1;
    b_size[1] = 1;
    colourSegmentation_kernel56<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_b_data);
    b_data_dirtyOnGpu = true;
    nz = 0;
    hipMemcpy(b_gpu_nz, &nz, 4UL, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b_size, &b_size[0], 8UL, hipMemcpyHostToDevice);
    colourSegmentation_kernel57<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (b_gpu_nz);
    hipMemcpy(&nz, b_gpu_nz, 4UL, hipMemcpyDeviceToHost);
  }

  seed_indices_size[0] = r_size[1];
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((r_size[1] - 1)
    + 1L), &k_grid, &k_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_r_size, &r_size[0], 8UL, hipMemcpyHostToDevice);
    colourSegmentation_kernel58<<<k_grid, k_block>>>(*gpu_c_data, *gpu_r_data,
      *gpu_r_size, *gpu_seed_indices_data);
  }

  colourSegmentation_kernel59<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_colourSegmentationMask, *gpu_SegmentationMask);
  locationsVar_size[0] = 0;
  if (seed_indices_size[0] != 0) {
    locationsVar_size[0] = 1;
    colourSegmentation_kernel60<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_seed_indices_data, *gpu_locationsVar_data);
    locationsVar_data_dirtyOnGpu = true;
  }

  badPixels_size[0] = static_cast<signed char>(seed_indices_size[0]);
  validLaunchParams = mwGetLaunchParameters(static_cast<double>
    ((seed_indices_size[0] - 1) + 1L), &l_grid, &l_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_seed_indices_size, &seed_indices_size[0], 4UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel61<<<l_grid, l_block>>>(*gpu_seed_indices_data,
      *gpu_seed_indices_size, *gpu_b_data);
    b_data_dirtyOnGpu = true;
  }

  validLaunchParams = false;
  nz = 1;
  exitg1 = false;
  while ((!exitg1) && (nz <= badPixels_size[0])) {
    if (b_data_dirtyOnGpu) {
      hipMemcpy(&b_data[0], gpu_b_data, 1UL, hipMemcpyDeviceToHost);
      b_data_dirtyOnGpu = false;
    }

    if (b_data[0]) {
      validLaunchParams = true;
      exitg1 = true;
    } else {
      nz = 2;
    }
  }

  if (validLaunchParams) {
    b_nz = static_cast<int>(((-1.0 - static_cast<double>(badPixels_size[0])) +
      1.0) / -1.0);
    for (i2 = 0; i2 < b_nz; i2++) {
      if (b_data_dirtyOnGpu) {
        hipMemcpy(&b_data[0], gpu_b_data, 1UL, hipMemcpyDeviceToHost);
        b_data_dirtyOnGpu = false;
      }

      if (b_data[0]) {
        nz = locationsVar_size[0];
        for (i = 0; i <= nz - 2; i++) {
          if (locationsVar_data_dirtyOnGpu) {
            hipMemcpy(&locationsVar_data[0], gpu_locationsVar_data, 4UL,
                       hipMemcpyDeviceToHost);
            locationsVar_data_dirtyOnGpu = false;
          }

          locationsVar_data[i] = locationsVar_data[i + 1];
          locationsVar_data_dirtyOnCpu = true;
        }

        locationsVar_size[0] = 0;
      }
    }
  }

  colourSegmentation_kernel62<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_SegmentationMask, *gpu_marker, *gpu_mask);
  validLaunchParams = mwGetLaunchParameters(static_cast<double>
    ((locationsVar_size[0] - 1) + 1L), &m_grid, &m_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (locationsVar_data_dirtyOnCpu) {
      hipMemcpy(gpu_locationsVar_data, &locationsVar_data[0], 4UL,
                 hipMemcpyHostToDevice);
    }

    hipMemcpy(gpu_locationsVar_size, &locationsVar_size[0], 4UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel63<<<m_grid, m_block>>>(*gpu_mask,
      *gpu_locationsVar_data, *gpu_locationsVar_size, *gpu_marker);
  }

  colourSegmentation_kernel64<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_marker, *gpu_marker_tmp);
  imreconstruct_cuda(&(*gpu_marker_tmp)[0], &(*gpu_mask)[0], 984U, 740U, 8UL, &(*
    gpu_marker)[0]);
  colourSegmentation_kernel65<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_colourSegmentationMask, *gpu_marker, *gpu_SegmentationMask);
  hipMemcpy(&SegmentationMask[0], gpu_SegmentationMask, 728160UL,
             hipMemcpyDeviceToHost);
  hipFree(*gpu_colourBalancedImage);
  hipFree(*gpu_Xin);
  hipFree(*gpu_HSVImage);
  hipFree(b_gpu_nz);
  hipFree(gpu_nz);
  hipFree(*gpu_HSVImage_size);
  hipFree(*gpu_centerColourMask);
  hipFree(*gpu_HSVImage_data);
  hipFree(*b_gpu_HSVImage_size);
  hipFree(*c_gpu_HSVImage_size);
  hipFree(*d_gpu_HSVImage_size);
  hipFree(*e_gpu_HSVImage_size);
  hipFree(*f_gpu_HSVImage_size);
  hipFree(*g_gpu_HSVImage_size);
  hipFree(*h_gpu_HSVImage_size);
  hipFree(*i_gpu_HSVImage_size);
  hipFree(*gpu_outputs);
  hipFree(*gpu_colourSegmentationMask);
  hipFree(*gpu_maskPad);
  hipFree(*b_gpu_mask);
  hipFree(*gpu_markerPad);
  hipFree(*gpu_idx);
  hipFree(*gpu_I1);
  hipFree(*gpu_x);
  hipFree(*gpu_r_size);
  hipFree(*gpu_b_size);
  hipFree(*gpu_b_data);
  hipFree(*gpu_r_data);
  hipFree(*gpu_c_data);
  hipFree(*gpu_seed_indices_size);
  hipFree(*gpu_locationsVar_size);
  hipFree(*gpu_seed_indices_data);
  hipFree(*gpu_locationsVar_data);
  hipFree(*gpu_SegmentationMask);
  hipFree(*gpu_mask);
  hipFree(*gpu_marker);
  hipFree(*gpu_marker_tmp);
}

//
// File trailer for colourSegmentation.cu
//
// [EOF]
//
