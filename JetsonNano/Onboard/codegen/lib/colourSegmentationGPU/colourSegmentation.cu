#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: colourSegmentation.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 04-Nov-2020 15:12:57
//

// Include Files
#include "colourSegmentation.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"

// Type Definitions
struct cell_wrap_7
{
  double f1[9];
};

// Function Declarations
static __global__ void colourSegmentation_kernel1(const unsigned char
  colourBalancedImage[2184480], double Xin[2184480]);
static __global__ void colourSegmentation_kernel10(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel11(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel12(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel13(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel14(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel15(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel16(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel17(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel18(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel19(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel2(const double Xin[2184480],
  double HSVImage[2184480]);
static __global__ void colourSegmentation_kernel20(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel21(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel22(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel23(const bool centerColourMask
  [23409], cell_wrap_7 outputs[1]);
static __global__ void colourSegmentation_kernel24(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel25(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel26(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel27(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel28(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel29(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel3(double HSVImage[2184480]);
static __global__ void colourSegmentation_kernel30(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel31(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel32(const double HSVImage[2184480],
  bool SegmentationMask[728160]);
static __global__ void colourSegmentation_kernel4(bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel5(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel6(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel7(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);
static __global__ void colourSegmentation_kernel8(const bool HSVImage_data
  [728160], bool centerColourMask[23409]);
static __global__ void colourSegmentation_kernel9(const double HSVImage[2184480],
  const int i2, const int i, const int HSVImage_size[2], const int b_i, const
  int i1, bool HSVImage_data[728160]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char colourBalancedImage[2184480]
//                double Xin[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel1(
  const unsigned char colourBalancedImage[2184480], double Xin[2184480])
{
  int ix;
  ix = static_cast<int>(mwGetGlobalThreadIndex());
  if (ix < 2184480) {
    // Input:
    // colourBalancedImage = RGB image 984x740x3
    // centerOfObjectX = Objects center postion in the x-axis
    // centerOfObjectY = Objects center postion in the y-axis
    // Output:
    // SegmentationMask = bitmask of the segmented object
    // Transform the Image into the HSV colour space
    Xin[ix] = static_cast<double>(colourBalancedImage[ix]) / 255.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel10(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 5202] = HSVImage_data[xpageoffset
      + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel11(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] > 70.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] < 155.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] == 70.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel12(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 7803] = HSVImage_data[xpageoffset
      + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel13(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] > 155.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] < 205.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] == 155.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel14(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 10404] =
      HSVImage_data[xpageoffset + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel15(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] > 205.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] < 260.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] == 205.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel16(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 13005] =
      HSVImage_data[xpageoffset + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel17(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] > 260.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] < 315.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] == 260.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel18(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 15606] =
      HSVImage_data[xpageoffset + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel19(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 1456320]
                       > 0.6)) || (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] == 0.6)))) && (static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] <
                       0.2)) || (static_cast<int>(HSVImage[((i + xpageoffset) +
      984 * (i2 + ix)) + 728160] == 0.2)))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Xin[2184480]
//                double HSVImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel2(
  const double Xin[2184480], double HSVImage[2184480])
{
  double varargin_1[3];
  double h;
  double tmp;
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    double d1;
    double d2;
    double delta;
    double v;
    int i;
    d = Xin[xpageoffset + 984 * ix];
    varargin_1[0] = d;
    d1 = Xin[(xpageoffset + 984 * ix) + 728160];
    varargin_1[1] = d1;
    d2 = Xin[(xpageoffset + 984 * ix) + 1456320];
    varargin_1[2] = d2;
    v = d;
    for (i = 0; i < 2; i++) {
      if (v < varargin_1[i + 1]) {
        v = varargin_1[i + 1];
      }
    }

    varargin_1[0] = d;
    varargin_1[1] = d1;
    varargin_1[2] = d2;
    tmp = d;
    for (i = 0; i < 2; i++) {
      if (tmp > varargin_1[i + 1]) {
        tmp = varargin_1[i + 1];
      }
    }

    delta = v - tmp;
    tmp = delta;
    if (delta == 0.0) {
      tmp = 1.0;
    }

    h = 0.0;
    if (d == v) {
      h = (d1 - d2) / tmp;
    }

    if (d1 == v) {
      h = (d2 - d) / tmp + 2.0;
    }

    if (d2 == v) {
      h = (d - d1) / tmp + 4.0;
    }

    h /= 6.0;
    if (h < 0.0) {
      h++;
    }

    tmp /= v;
    if (delta == 0.0) {
      h = 0.0;
      tmp = 0.0;
    }

    if (!static_cast<int>(v != 0.0)) {
      tmp = 0.0;
    }

    HSVImage[xpageoffset + 984 * ix] = h;
    HSVImage[(xpageoffset + 984 * ix) + 728160] = tmp;
    HSVImage[(xpageoffset + 984 * ix) + 1456320] = v;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel20(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 18207] =
      HSVImage_data[xpageoffset + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel21(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 1456320]
                       < 0.3)) || (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] == 0.3)))) || (static_cast<int>((
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 1456320]
                       < 0.6)) && (static_cast<int>((static_cast<int>(HSVImage
      [((i + xpageoffset) + 984 * (i2 + ix)) + 728160] < 0.2)) || (static_cast<
      int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] == 0.2)))))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel22(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 20808] =
      HSVImage_data[xpageoffset + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool centerColourMask[23409]
//                cell_wrap_7 outputs[1]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourSegmentation_kernel23(
  const bool centerColourMask[23409], cell_wrap_7 outputs[1])
{
  double d;
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 9) {
    int xpageoffset;

    //  Sum up all the masks and then determine which mask has the most pixels
    xpageoffset = i * 2601;
    d = static_cast<double>(centerColourMask[xpageoffset]);
    for (int ix = 0; ix < 2600; ix++) {
      d += static_cast<double>(centerColourMask[(xpageoffset + ix) + 1]);
    }

    outputs[0].f1[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel24(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    double d1;
    d = HSVImage[(xpageoffset + 984 * ix) + 1456320];
    d1 = HSVImage[(xpageoffset + 984 * ix) + 728160];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>(d < 0.3)) || (static_cast<int>(d == 0.3)))) || (static_cast<int>((
      static_cast<int>(d < 0.6)) && (static_cast<int>((static_cast<int>(d1 < 0.2))
      || (static_cast<int>(d1 == 0.2)))))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel25(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    double d1;
    d = HSVImage[(xpageoffset + 984 * ix) + 1456320];
    d1 = HSVImage[(xpageoffset + 984 * ix) + 728160];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>(d > 0.6)) || (static_cast<int>(d == 0.6)))) && (static_cast<int>((
      static_cast<int>(d1 < 0.2)) || (static_cast<int>(d1 == 0.2)))));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel26(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d > 260.0)) && (static_cast<int>
      (d < 315.0)))) || (static_cast<int>(d == 260.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel27(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d > 205.0)) && (static_cast<int>
      (d < 260.0)))) || (static_cast<int>(d == 205.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel28(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d > 155.0)) && (static_cast<int>
      (d < 205.0)))) || (static_cast<int>(d == 155.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel29(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d > 70.0)) && (static_cast<int>(d
      < 155.0)))) || (static_cast<int>(d == 70.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double HSVImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel3
  (double HSVImage[2184480])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    HSVImage[xpageoffset + 984 * ix] *= 360.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel30(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d > 50.0)) && (static_cast<int>(d
      < 70.0)))) || (static_cast<int>(d == 50.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel31(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d > 15.0)) && (static_cast<int>(d
      < 50.0)))) || (static_cast<int>(d == 15.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                bool SegmentationMask[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel32(
  const double HSVImage[2184480], bool SegmentationMask[728160])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 984UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        984UL);
  if ((static_cast<int>(ix < 740)) && (static_cast<int>(xpageoffset < 984))) {
    double d;
    d = HSVImage[xpageoffset + 984 * ix];
    SegmentationMask[xpageoffset + 984 * ix] = ((static_cast<int>((static_cast<
      int>((static_cast<int>((static_cast<int>(d < 15.0)) || (static_cast<int>(d
      == 315.0)))) || (static_cast<int>(d > 315.0)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 728160] > 0.2)))) && (static_cast<int>
      (HSVImage[(xpageoffset + 984 * ix) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel4(bool
  centerColourMask[23409])
{
  int ix;
  ix = static_cast<int>(mwGetGlobalThreadIndex());
  if (ix < 23409) {
    centerColourMask[ix] = false;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel5(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] < 15.0)) || (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] == 315.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] > 315.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel6(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[xpageoffset + 51 * ix] = HSVImage_data[xpageoffset + 51 *
      ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel7(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] > 15.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] < 50.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] == 15.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const bool HSVImage_data[728160]
//                bool centerColourMask[23409]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourSegmentation_kernel8(
  const bool HSVImage_data[728160], bool centerColourMask[23409])
{
  unsigned long threadId;
  int ix;
  int xpageoffset;
  threadId = mwGetGlobalThreadIndex();
  xpageoffset = static_cast<int>(threadId % 51UL);
  ix = static_cast<int>((threadId - static_cast<unsigned long>(xpageoffset)) /
                        51UL);
  if ((static_cast<int>(ix < 51)) && (static_cast<int>(xpageoffset < 51))) {
    centerColourMask[(xpageoffset + 51 * ix) + 2601] = HSVImage_data[xpageoffset
      + 51 * ix];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double HSVImage[2184480]
//                const int i2
//                const int i
//                const int HSVImage_size[2]
//                const int b_i
//                const int i1
//                bool HSVImage_data[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void colourSegmentation_kernel9(
  const double HSVImage[2184480], const int i2, const int i, const int
  HSVImage_size[2], const int b_i, const int i1, bool HSVImage_data[728160])
{
  unsigned long idx;
  long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<long>(b_i) + 1L) * (static_cast<long>(i1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<unsigned long>(loopEnd); idx +=
       threadStride) {
    int ix;
    int xpageoffset;
    xpageoffset = static_cast<int>(idx % (static_cast<unsigned long>(b_i) + 1UL));
    ix = static_cast<int>((idx - static_cast<unsigned long>(xpageoffset)) / (
      static_cast<unsigned long>(b_i) + 1UL));
    HSVImage_data[xpageoffset + HSVImage_size[0] * ix] = ((static_cast<int>((
      static_cast<int>((static_cast<int>((static_cast<int>(HSVImage[(i +
      xpageoffset) + 984 * (i2 + ix)] > 50.0)) && (static_cast<int>(HSVImage[(i
      + xpageoffset) + 984 * (i2 + ix)] < 70.0)))) || (static_cast<int>
      (HSVImage[(i + xpageoffset) + 984 * (i2 + ix)] == 50.0)))) && (
      static_cast<int>(HSVImage[((i + xpageoffset) + 984 * (i2 + ix)) + 728160] >
                       0.2)))) && (static_cast<int>(HSVImage[((i + xpageoffset)
      + 984 * (i2 + ix)) + 1456320] > 0.3)));
  }
}

//
// The function takes in a image and a rough postion of the object in the
// image that should be segmented. and it outputs a a bitmask of the the
// object.
// Arguments    : const unsigned char colourBalancedImage[2184480]
//                double centerOfObjectX
//                double centerOfObjectY
//                bool SegmentationMask[728160]
// Return Type  : void
//
void colourSegmentation(const unsigned char colourBalancedImage[2184480], double
  centerOfObjectX, double centerOfObjectY, bool SegmentationMask[728160])
{
  cell_wrap_7 outputs[1];
  cell_wrap_7 (*gpu_outputs)[1];
  dim3 b_block;
  dim3 b_grid;
  dim3 block;
  dim3 c_block;
  dim3 c_grid;
  dim3 d_block;
  dim3 d_grid;
  dim3 e_block;
  dim3 e_grid;
  dim3 f_block;
  dim3 f_grid;
  dim3 g_block;
  dim3 g_grid;
  dim3 grid;
  dim3 h_block;
  dim3 h_grid;
  dim3 i_block;
  dim3 i_grid;
  double (*gpu_HSVImage)[2184480];
  double (*gpu_Xin)[2184480];
  double ex;
  int HSVImage_size[2];
  int b_HSVImage_size[2];
  int c_HSVImage_size[2];
  int d_HSVImage_size[2];
  int e_HSVImage_size[2];
  int f_HSVImage_size[2];
  int g_HSVImage_size[2];
  int h_HSVImage_size[2];
  int i_HSVImage_size[2];
  int (*b_gpu_HSVImage_size)[2];
  int (*c_gpu_HSVImage_size)[2];
  int (*d_gpu_HSVImage_size)[2];
  int (*e_gpu_HSVImage_size)[2];
  int (*f_gpu_HSVImage_size)[2];
  int (*g_gpu_HSVImage_size)[2];
  int (*gpu_HSVImage_size)[2];
  int (*h_gpu_HSVImage_size)[2];
  int (*i_gpu_HSVImage_size)[2];
  int b_i;
  int b_i1;
  int i;
  int i1;
  int i2;
  int i3;
  unsigned char (*gpu_colourBalancedImage)[2184480];
  bool (*gpu_HSVImage_data)[728160];
  bool (*gpu_SegmentationMask)[728160];
  bool (*gpu_centerColourMask)[23409];
  bool validLaunchParams;
  hipMalloc(&gpu_SegmentationMask, 728160UL);
  hipMalloc(&gpu_outputs, 72UL);
  hipMalloc(&i_gpu_HSVImage_size, 8UL);
  hipMalloc(&h_gpu_HSVImage_size, 8UL);
  hipMalloc(&g_gpu_HSVImage_size, 8UL);
  hipMalloc(&f_gpu_HSVImage_size, 8UL);
  hipMalloc(&e_gpu_HSVImage_size, 8UL);
  hipMalloc(&d_gpu_HSVImage_size, 8UL);
  hipMalloc(&c_gpu_HSVImage_size, 8UL);
  hipMalloc(&b_gpu_HSVImage_size, 8UL);
  hipMalloc(&gpu_HSVImage_data, 728160UL);
  hipMalloc(&gpu_centerColourMask, 23409UL);
  hipMalloc(&gpu_HSVImage_size, 8UL);
  hipMalloc(&gpu_HSVImage, 17475840UL);
  hipMalloc(&gpu_Xin, 17475840UL);
  hipMalloc(&gpu_colourBalancedImage, 2184480UL);

  // Input:
  // colourBalancedImage = RGB image 984x740x3
  // centerOfObjectX = Objects center postion in the x-axis
  // centerOfObjectY = Objects center postion in the y-axis
  // Output:
  // SegmentationMask = bitmask of the segmented object
  // Transform the Image into the HSV colour space
  hipMemcpy(gpu_colourBalancedImage, (void *)&colourBalancedImage[0], 2184480UL,
             hipMemcpyHostToDevice);
  colourSegmentation_kernel1<<<dim3(4267U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_colourBalancedImage, *gpu_Xin);
  colourSegmentation_kernel2<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_Xin, *gpu_HSVImage);

  // Standardize the H-values from 0-1 to 0-360
  colourSegmentation_kernel3<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage);

  // Make a small square at the center of the object and determine which
  // colour is most prevalent in the square
  // Choose square size
  // Calculate the start and end values for X and  Y
  // Take out the sqaure area from the image
  if (centerOfObjectX - 25.0 > centerOfObjectX + 25.0) {
    i = 0;
    i1 = 0;
  } else {
    i = static_cast<int>(centerOfObjectX - 25.0) - 1;
    i1 = static_cast<int>(centerOfObjectX + 25.0);
  }

  if (centerOfObjectY - 25.0 > centerOfObjectY + 25.0) {
    i2 = 0;
    i3 = 0;
  } else {
    i2 = static_cast<int>(centerOfObjectY - 25.0) - 1;
    i3 = static_cast<int>(centerOfObjectY + 25.0);
  }

  colourSegmentation_kernel4<<<dim3(46U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_centerColourMask);

  // Make mask in the square for every 9 colours.
  // Calcualte which pixels are red
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  HSVImage_size[0] = b_i + 1;
  HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_HSVImage_size, &HSVImage_size[0], 8UL, hipMemcpyHostToDevice);
    colourSegmentation_kernel5<<<grid, block>>>(*gpu_HSVImage, i2, i,
      *gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel6<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are orange
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  b_HSVImage_size[0] = b_i + 1;
  b_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &b_grid, &b_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(b_gpu_HSVImage_size, &b_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel7<<<b_grid, b_block>>>(*gpu_HSVImage, i2, i,
      *b_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel8<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are yellow
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  c_HSVImage_size[0] = b_i + 1;
  c_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &c_grid, &c_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(c_gpu_HSVImage_size, &c_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel9<<<c_grid, c_block>>>(*gpu_HSVImage, i2, i,
      *c_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel10<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are green
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  d_HSVImage_size[0] = b_i + 1;
  d_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &d_grid, &d_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(d_gpu_HSVImage_size, &d_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel11<<<d_grid, d_block>>>(*gpu_HSVImage, i2, i,
      *d_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel12<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are cyan
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  e_HSVImage_size[0] = b_i + 1;
  e_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &e_grid, &e_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(e_gpu_HSVImage_size, &e_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel13<<<e_grid, e_block>>>(*gpu_HSVImage, i2, i,
      *e_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel14<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are blue
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  f_HSVImage_size[0] = b_i + 1;
  f_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &f_grid, &f_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(f_gpu_HSVImage_size, &f_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel15<<<f_grid, f_block>>>(*gpu_HSVImage, i2, i,
      *f_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel16<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are purple
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  g_HSVImage_size[0] = b_i + 1;
  g_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &g_grid, &g_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(g_gpu_HSVImage_size, &g_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel17<<<g_grid, g_block>>>(*gpu_HSVImage, i2, i,
      *g_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel18<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are white
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  h_HSVImage_size[0] = b_i + 1;
  h_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &h_grid, &h_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(h_gpu_HSVImage_size, &h_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel19<<<h_grid, h_block>>>(*gpu_HSVImage, i2, i,
      *h_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel20<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  // Calcualte which pixels are black
  b_i = (i1 - i) - 1;
  b_i1 = (i3 - i2) - 1;
  i_HSVImage_size[0] = b_i + 1;
  i_HSVImage_size[1] = b_i1 + 1;
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((b_i + 1L) *
    (b_i1 + 1L)), &i_grid, &i_block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(i_gpu_HSVImage_size, &i_HSVImage_size[0], 8UL,
               hipMemcpyHostToDevice);
    colourSegmentation_kernel21<<<i_grid, i_block>>>(*gpu_HSVImage, i2, i,
      *i_gpu_HSVImage_size, b_i, b_i1, *gpu_HSVImage_data);
  }

  colourSegmentation_kernel22<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_HSVImage_data, *gpu_centerColourMask);

  //  Sum up all the masks and then determine which mask has the most pixels
  colourSegmentation_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_centerColourMask, *gpu_outputs);
  i1 = 1;
  hipMemcpy(&outputs[0], gpu_outputs, 72UL, hipMemcpyDeviceToHost);
  validLaunchParams = false;
  ex = outputs[0].f1[0];
  for (i = 0; i < 8; i++) {
    if (validLaunchParams) {
      hipMemcpy(&outputs[0], gpu_outputs, 72UL, hipMemcpyDeviceToHost);
      validLaunchParams = false;
    }

    if (ex < outputs[0].f1[i + 1]) {
      ex = outputs[0].f1[i + 1];
      i1 = i + 2;
    }
  }

  //  Make a mask over the whole image. The colour that is seperated
  //  is the one that had the most pixels in the square
  if (i1 == 1) {
    // Red
    colourSegmentation_kernel32<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 2) {
    // Orange
    colourSegmentation_kernel31<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 3) {
    // Yellow
    colourSegmentation_kernel30<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 4) {
    // Green
    colourSegmentation_kernel29<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 5) {
    // Cyan
    colourSegmentation_kernel28<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 6) {
    // Blue
    colourSegmentation_kernel27<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 7) {
    // Purple
    colourSegmentation_kernel26<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else if (i1 == 8) {
    // White
    colourSegmentation_kernel25<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  } else {
    // Black
    colourSegmentation_kernel24<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_HSVImage, *gpu_SegmentationMask);
  }

  // Fill up the hole in the binary image.
  // maskFill =imfill(colourSegmentationMask,'holes');
  //
  //  SegmentationMask = bwselect(maskFill,centerOfObjectY,centerOfObjectX);
  hipMemcpy(&SegmentationMask[0], gpu_SegmentationMask, 728160UL,
             hipMemcpyDeviceToHost);
  hipFree(*gpu_colourBalancedImage);
  hipFree(*gpu_Xin);
  hipFree(*gpu_HSVImage);
  hipFree(*gpu_HSVImage_size);
  hipFree(*gpu_centerColourMask);
  hipFree(*gpu_HSVImage_data);
  hipFree(*b_gpu_HSVImage_size);
  hipFree(*c_gpu_HSVImage_size);
  hipFree(*d_gpu_HSVImage_size);
  hipFree(*e_gpu_HSVImage_size);
  hipFree(*f_gpu_HSVImage_size);
  hipFree(*g_gpu_HSVImage_size);
  hipFree(*h_gpu_HSVImage_size);
  hipFree(*i_gpu_HSVImage_size);
  hipFree(*gpu_outputs);
  hipFree(*gpu_SegmentationMask);
}

//
// File trailer for colourSegmentation.cu
//
// [EOF]
//
