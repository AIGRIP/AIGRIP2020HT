#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: colourBalance.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 19-Nov-2020 17:09:04
//

// Include Files
#include "colourBalance.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __device__ double atomicOpreal_T(double *address, double value);
static __global__ void colourBalance_kernel1(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel2(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel3(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel4(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel5(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel6(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel7(const double *y, const double
  meanRed, const unsigned char originalImage[2184480], unsigned char
  newGreenLayer[728160], unsigned char newRedLayer[728160]);
static __global__ void colourBalance_kernel8(const double meanRed, const
  unsigned char originalImage[2184480], unsigned char newBlueLayer[728160]);
static __global__ void colourBalance_kernel9(const unsigned char newRedLayer
  [728160], const int initAuxVar, unsigned char colourBalancedImage[2184480]);
static __device__ double rt_roundd_snf_device(double u);
static __device__ double shflDown2(double in1, unsigned int offset, unsigned int
  mask);
static __device__ double threadGroupReduction(double val, unsigned int lane,
  unsigned int mask);
static __device__ double workGroupReduction(double val, unsigned int mask,
  unsigned int numActiveWarps);

// Function Definitions
//
// Arguments    : double *address
//                double value
// Return Type  : double
//
static __device__ double atomicOpreal_T(double *address, double value)
{
  unsigned long long int old;
  unsigned long long int *address_as_up;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old, __double_as_longlong(value +
      __longlong_as_double(old)));
  } while (assumed != old);

  return __longlong_as_double(old);
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourBalance_kernel1(const
  unsigned char originalImage[2184480], double *y)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    // Input:
    // originalImage = RGB image 984x740x3
    // Output:
    // colourBalancedImage = RGB image 984x740x3
    // Calculate number of pixales in te image
    // Calculate the mean of every colour and take the mean of every colour added 
    // togheter
    *y = static_cast<double>(originalImage[0]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel2(const
  unsigned char originalImage[2184480], double *y)
{
  double tmpRed0;
  unsigned int blockStride;
  unsigned int idx;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  if (mwIsLastBlock()) {
    m = 728159U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 728158U) {
    tmpRed0 = static_cast<double>(originalImage[(static_cast<int>(threadId) + 1)
      % 984 + 984 * ((static_cast<int>(threadId) + 1) / 984)]);
  }

  m = __ballot_sync(MAX_uint32_T, threadId <= 728158U);
  for (idx = threadId + threadStride; idx <= 728158U; idx += threadStride) {
    tmpRed0 += static_cast<double>(originalImage[(static_cast<int>(idx) + 1) %
      984 + 984 * ((static_cast<int>(idx) + 1) / 984)]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourBalance_kernel3(const
  unsigned char originalImage[2184480], double *y)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *y = static_cast<double>(originalImage[728160]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel4(const
  unsigned char originalImage[2184480], double *y)
{
  double tmpRed0;
  unsigned int blockStride;
  unsigned int idx;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  if (mwIsLastBlock()) {
    m = 728159U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 728158U) {
    tmpRed0 = static_cast<double>(originalImage[((static_cast<int>(threadId) + 1)
      % 984 + 984 * ((static_cast<int>(threadId) + 1) / 984)) + 728160]);
  }

  m = __ballot_sync(MAX_uint32_T, threadId <= 728158U);
  for (idx = threadId + threadStride; idx <= 728158U; idx += threadStride) {
    tmpRed0 += static_cast<double>(originalImage[((static_cast<int>(idx) + 1) %
      984 + 984 * ((static_cast<int>(idx) + 1) / 984)) + 728160]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourBalance_kernel5(const
  unsigned char originalImage[2184480], double *y)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *y = static_cast<double>(originalImage[1456320]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel6(const
  unsigned char originalImage[2184480], double *y)
{
  double tmpRed0;
  unsigned int blockStride;
  unsigned int idx;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  if (mwIsLastBlock()) {
    m = 728159U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 728158U) {
    tmpRed0 = static_cast<double>(originalImage[((static_cast<int>(threadId) + 1)
      % 984 + 984 * ((static_cast<int>(threadId) + 1) / 984)) + 1456320]);
  }

  m = __ballot_sync(MAX_uint32_T, threadId <= 728158U);
  for (idx = threadId + threadStride; idx <= 728158U; idx += threadStride) {
    tmpRed0 += static_cast<double>(originalImage[((static_cast<int>(idx) + 1) %
      984 + 984 * ((static_cast<int>(idx) + 1) / 984)) + 1456320]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double *y
//                const double meanRed
//                const unsigned char originalImage[2184480]
//                unsigned char newGreenLayer[728160]
//                unsigned char newRedLayer[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel7(const
  double *y, const double meanRed, const unsigned char originalImage[2184480],
  unsigned char newGreenLayer[728160], unsigned char newRedLayer[728160])
{
  unsigned long threadId;
  int i;
  int k;
  threadId = mwGetGlobalThreadIndex();
  i = static_cast<int>(threadId % 984UL);
  k = static_cast<int>((threadId - static_cast<unsigned long>(i)) / 984UL);
  if ((static_cast<int>(k < 740)) && (static_cast<int>(i < 984))) {
    double d;
    unsigned char u;
    d = rt_roundd_snf_device(static_cast<double>(originalImage[i + 984 * k]) *
      meanRed);
    if (d < 256.0) {
      u = static_cast<unsigned char>(d);
    } else if (d >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    newRedLayer[i + 984 * k] = u;
    d = rt_roundd_snf_device(static_cast<double>(originalImage[(i + 984 * k) +
      728160]) * *y);
    if (d < 256.0) {
      u = static_cast<unsigned char>(d);
    } else if (d >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    newGreenLayer[i + 984 * k] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double meanRed
//                const unsigned char originalImage[2184480]
//                unsigned char newBlueLayer[728160]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel8(const
  double meanRed, const unsigned char originalImage[2184480], unsigned char
  newBlueLayer[728160])
{
  unsigned long threadId;
  int i;
  int k;
  threadId = mwGetGlobalThreadIndex();
  i = static_cast<int>(threadId % 984UL);
  k = static_cast<int>((threadId - static_cast<unsigned long>(i)) / 984UL);
  if ((static_cast<int>(k < 740)) && (static_cast<int>(i < 984))) {
    double d;
    unsigned char u;
    d = rt_roundd_snf_device(static_cast<double>(originalImage[(i + 984 * k) +
      1456320]) * meanRed);
    if (d < 256.0) {
      u = static_cast<unsigned char>(d);
    } else if (d >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    newBlueLayer[i + 984 * k] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char newRedLayer[728160]
//                const int initAuxVar
//                unsigned char colourBalancedImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel9(const
  unsigned char newRedLayer[728160], const int initAuxVar, unsigned char
  colourBalancedImage[2184480])
{
  int j;
  j = static_cast<int>(mwGetGlobalThreadIndex());
  if (j < 728160) {
    colourBalancedImage[(initAuxVar + j) + 1] = newRedLayer[j];
  }
}

//
// Arguments    : double u
// Return Type  : double
//
static __device__ double rt_roundd_snf_device(double u)
{
  double y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

//
// Arguments    : double in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : double
//
static __device__ double shflDown2(double in1, unsigned int offset, unsigned int
  mask)
{
  int2 tmp;
  tmp = *(int2 *)&in1;
  tmp.x = __shfl_down_sync(mask, tmp.x, offset);
  tmp.y = __shfl_down_sync(mask, tmp.y, offset);
  return *(double *)&tmp;
}

//
// Arguments    : double val
//                unsigned int lane
//                unsigned int mask
// Return Type  : double
//
static __device__ double threadGroupReduction(double val, unsigned int lane,
  unsigned int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    double other;
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }

    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }

  return val;
}

//
// Arguments    : double val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : double
//
static __device__ double workGroupReduction(double val, unsigned int mask,
  unsigned int numActiveWarps)
{
  __shared__ double shared[32];
  unsigned int lane;
  unsigned int thBlkId;
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }

  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }

  return val;
}

//
// The function takes in a image and balances the colours. So that different
// lightnings will not have that big effect.
// Arguments    : const unsigned char originalImage[2184480]
//                unsigned char colourBalancedImage[2184480]
// Return Type  : void
//
void colourBalance(const unsigned char originalImage[2184480], unsigned char
                   colourBalancedImage[2184480])
{
  static unsigned char newBlueLayer[728160];
  static unsigned char newGreenLayer[728160];
  double meanBlue;
  double meanGreen;
  double meanRed;
  double meanTotal;
  double y;
  double *gpu_y;
  int initAuxVar;
  int iy;
  int j;
  unsigned char (*gpu_colourBalancedImage)[2184480];
  unsigned char (*gpu_originalImage)[2184480];
  unsigned char (*gpu_newBlueLayer)[728160];
  unsigned char (*gpu_newGreenLayer)[728160];
  unsigned char (*gpu_newRedLayer)[728160];
  bool colourBalancedImage_dirtyOnCpu;
  bool newBlueLayer_dirtyOnGpu;
  bool originalImage_dirtyOnCpu;
  hipMalloc(&gpu_newRedLayer, 728160UL);
  hipMalloc(&gpu_newGreenLayer, 728160UL);
  hipMalloc(&gpu_colourBalancedImage, 2184480UL);
  hipMalloc(&gpu_newBlueLayer, 728160UL);
  hipMalloc(&gpu_y, 8UL);
  hipMalloc(&gpu_originalImage, 2184480UL);
  colourBalancedImage_dirtyOnCpu = false;

  // Input:
  // originalImage = RGB image 984x740x3
  // Output:
  // colourBalancedImage = RGB image 984x740x3
  // Calculate number of pixales in te image
  // Calculate the mean of every colour and take the mean of every colour added
  // togheter
  hipMemcpy(gpu_originalImage, (void *)&originalImage[0], 2184480UL,
             hipMemcpyHostToDevice);
  colourBalance_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  colourBalance_kernel2<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  hipMemcpy(&y, gpu_y, 8UL, hipMemcpyDeviceToHost);
  meanRed = y / 728160.0;
  colourBalance_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  colourBalance_kernel4<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  hipMemcpy(&y, gpu_y, 8UL, hipMemcpyDeviceToHost);
  meanGreen = y / 728160.0;
  colourBalance_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  colourBalance_kernel6<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  hipMemcpy(&y, gpu_y, 8UL, hipMemcpyDeviceToHost);
  meanBlue = y / 728160.0;
  meanTotal = ((meanRed + meanGreen) + meanBlue) / 3.0;

  // Create a new image and use the mean value of the colours to calculate the
  // new colours
  meanRed = meanTotal / meanRed;
  y = meanTotal / meanGreen;
  hipMemcpy(gpu_y, &y, 8UL, hipMemcpyHostToDevice);
  colourBalance_kernel7<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_y,
    meanRed, *gpu_originalImage, *gpu_newGreenLayer, *gpu_newRedLayer);
  originalImage_dirtyOnCpu = true;
  colourBalance_kernel8<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>(meanTotal /
    meanBlue, *gpu_originalImage, *gpu_newBlueLayer);
  newBlueLayer_dirtyOnGpu = true;
  iy = -1;
  for (j = 0; j < 728160; j++) {
    iy = j;
    if (newBlueLayer_dirtyOnGpu) {
      hipMemcpy(&newBlueLayer[0], gpu_newBlueLayer, 728160UL,
                 hipMemcpyDeviceToHost);
      newBlueLayer_dirtyOnGpu = false;
    }

    colourBalancedImage[j] = newBlueLayer[j];
    colourBalancedImage_dirtyOnCpu = true;
  }

  initAuxVar = iy;
  for (j = 0; j < 728160; j++) {
    iy = (initAuxVar + j) + 1;
    if (originalImage_dirtyOnCpu) {
      hipMemcpy(&newGreenLayer[0], gpu_newGreenLayer, 728160UL,
                 hipMemcpyDeviceToHost);
      originalImage_dirtyOnCpu = false;
    }

    colourBalancedImage[iy] = newGreenLayer[j];
    colourBalancedImage_dirtyOnCpu = true;
  }

  if (colourBalancedImage_dirtyOnCpu) {
    hipMemcpy(gpu_colourBalancedImage, &colourBalancedImage[0], 2184480UL,
               hipMemcpyHostToDevice);
  }

  colourBalance_kernel9<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_newRedLayer, iy, *gpu_colourBalancedImage);
  hipMemcpy(&colourBalancedImage[0], gpu_colourBalancedImage, 2184480UL,
             hipMemcpyDeviceToHost);
  hipFree(*gpu_originalImage);
  hipFree(gpu_y);
  hipFree(*gpu_newBlueLayer);
  hipFree(*gpu_colourBalancedImage);
  hipFree(*gpu_newGreenLayer);
  hipFree(*gpu_newRedLayer);
}

//
// File trailer for colourBalance.cu
//
// [EOF]
//
