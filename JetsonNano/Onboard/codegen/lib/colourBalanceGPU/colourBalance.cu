#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: colourBalance.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 04-Nov-2020 10:52:24
//

// Include Files
#include "colourBalance.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __device__ double atomicOpreal_T(double *address, double value);
static __global__ void colourBalance_kernel1(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel2(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel3(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel4(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel5(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel6(const unsigned char originalImage
  [2184480], double *y);
static __global__ void colourBalance_kernel7(const double *y, const unsigned
  char originalImage[2184480], const double meanRed, unsigned char
  colourBalancedImage[2184480]);
static __global__ void colourBalance_kernel8(const unsigned char originalImage
  [2184480], const double meanRed, unsigned char colourBalancedImage[2184480]);
static __device__ double rt_roundd_snf_device(double u);
static __device__ double shflDown2(double in1, unsigned int offset, unsigned int
  mask);
static __device__ double threadGroupReduction(double val, unsigned int lane,
  unsigned int mask);
static __device__ double workGroupReduction(double val, unsigned int mask,
  unsigned int numActiveWarps);

// Function Definitions
//
// Arguments    : double *address
//                double value
// Return Type  : double
//
static __device__ double atomicOpreal_T(double *address, double value)
{
  unsigned long long int old;
  unsigned long long int *address_as_up;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old, __double_as_longlong(value +
      __longlong_as_double(old)));
  } while (assumed != old);

  return __longlong_as_double(old);
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourBalance_kernel1(const
  unsigned char originalImage[2184480], double *y)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    // Input:
    // originalImage = RGB image 984x740x3
    // Output:
    // colourBalancedImage = RGB image 984x740x3
    // Calculate number of pixales in te image
    // Calculate the mean of every colour and take the mean of every colour added 
    // togheter
    *y = static_cast<double>(originalImage[0]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel2(const
  unsigned char originalImage[2184480], double *y)
{
  double tmpRed0;
  unsigned int blockStride;
  unsigned int idx;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  if (mwIsLastBlock()) {
    m = 728159U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 728158U) {
    tmpRed0 = static_cast<double>(originalImage[(static_cast<int>(threadId) + 1)
      % 984 + 984 * ((static_cast<int>(threadId) + 1) / 984)]);
  }

  m = __ballot_sync(MAX_uint32_T, threadId <= 728158U);
  for (idx = threadId + threadStride; idx <= 728158U; idx += threadStride) {
    tmpRed0 += static_cast<double>(originalImage[(static_cast<int>(idx) + 1) %
      984 + 984 * ((static_cast<int>(idx) + 1) / 984)]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourBalance_kernel3(const
  unsigned char originalImage[2184480], double *y)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *y = static_cast<double>(originalImage[728160]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel4(const
  unsigned char originalImage[2184480], double *y)
{
  double tmpRed0;
  unsigned int blockStride;
  unsigned int idx;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  if (mwIsLastBlock()) {
    m = 728159U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 728158U) {
    tmpRed0 = static_cast<double>(originalImage[((static_cast<int>(threadId) + 1)
      % 984 + 984 * ((static_cast<int>(threadId) + 1) / 984)) + 728160]);
  }

  m = __ballot_sync(MAX_uint32_T, threadId <= 728158U);
  for (idx = threadId + threadStride; idx <= 728158U; idx += threadStride) {
    tmpRed0 += static_cast<double>(originalImage[((static_cast<int>(idx) + 1) %
      984 + 984 * ((static_cast<int>(idx) + 1) / 984)) + 728160]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void colourBalance_kernel5(const
  unsigned char originalImage[2184480], double *y)
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *y = static_cast<double>(originalImage[1456320]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                double *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel6(const
  unsigned char originalImage[2184480], double *y)
{
  double tmpRed0;
  unsigned int blockStride;
  unsigned int idx;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  if (mwIsLastBlock()) {
    m = 728159U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 728158U) {
    tmpRed0 = static_cast<double>(originalImage[((static_cast<int>(threadId) + 1)
      % 984 + 984 * ((static_cast<int>(threadId) + 1) / 984)) + 1456320]);
  }

  m = __ballot_sync(MAX_uint32_T, threadId <= 728158U);
  for (idx = threadId + threadStride; idx <= 728158U; idx += threadStride) {
    tmpRed0 += static_cast<double>(originalImage[((static_cast<int>(idx) + 1) %
      984 + 984 * ((static_cast<int>(idx) + 1) / 984)) + 1456320]);
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double *y
//                const unsigned char originalImage[2184480]
//                const double meanRed
//                unsigned char colourBalancedImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel7(const
  double *y, const unsigned char originalImage[2184480], const double meanRed,
  unsigned char colourBalancedImage[2184480])
{
  unsigned long threadId;
  int i;
  int k;
  threadId = mwGetGlobalThreadIndex();
  i = static_cast<int>(threadId % 984UL);
  k = static_cast<int>((threadId - static_cast<unsigned long>(i)) / 984UL);
  if ((static_cast<int>(k < 740)) && (static_cast<int>(i < 984))) {
    double d;
    unsigned char u;
    d = rt_roundd_snf_device(meanRed * static_cast<double>(originalImage[i + 984
      * k]));
    if (d < 256.0) {
      u = static_cast<unsigned char>(d);
    } else if (d >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    colourBalancedImage[i + 984 * k] = u;
    d = rt_roundd_snf_device(*y * static_cast<double>(originalImage[(i + 984 * k)
      + 728160]));
    if (d < 256.0) {
      u = static_cast<unsigned char>(d);
    } else if (d >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    colourBalancedImage[(i + 984 * k) + 728160] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char originalImage[2184480]
//                const double meanRed
//                unsigned char colourBalancedImage[2184480]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void colourBalance_kernel8(const
  unsigned char originalImage[2184480], const double meanRed, unsigned char
  colourBalancedImage[2184480])
{
  unsigned long threadId;
  int i;
  int k;
  threadId = mwGetGlobalThreadIndex();
  i = static_cast<int>(threadId % 984UL);
  k = static_cast<int>((threadId - static_cast<unsigned long>(i)) / 984UL);
  if ((static_cast<int>(k < 740)) && (static_cast<int>(i < 984))) {
    double d;
    unsigned char u;
    d = rt_roundd_snf_device(meanRed * static_cast<double>(originalImage[(i +
      984 * k) + 1456320]));
    if (d < 256.0) {
      u = static_cast<unsigned char>(d);
    } else if (d >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }

    colourBalancedImage[(i + 984 * k) + 1456320] = u;
  }
}

//
// Arguments    : double u
// Return Type  : double
//
static __device__ double rt_roundd_snf_device(double u)
{
  double y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

//
// Arguments    : double in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : double
//
static __device__ double shflDown2(double in1, unsigned int offset, unsigned int
  mask)
{
  int2 tmp;
  tmp = *(int2 *)&in1;
  tmp.x = __shfl_down_sync(mask, tmp.x, offset);
  tmp.y = __shfl_down_sync(mask, tmp.y, offset);
  return *(double *)&tmp;
}

//
// Arguments    : double val
//                unsigned int lane
//                unsigned int mask
// Return Type  : double
//
static __device__ double threadGroupReduction(double val, unsigned int lane,
  unsigned int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    double other;
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }

    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }

  return val;
}

//
// Arguments    : double val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : double
//
static __device__ double workGroupReduction(double val, unsigned int mask,
  unsigned int numActiveWarps)
{
  __shared__ double shared[32];
  unsigned int lane;
  unsigned int thBlkId;
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }

  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }

  return val;
}

//
// The function takes in a image and balances the colours. So that different
// lightnings will not have that big effect.
// Arguments    : const unsigned char originalImage[2184480]
//                unsigned char colourBalancedImage[2184480]
// Return Type  : void
//
void colourBalance(const unsigned char originalImage[2184480], unsigned char
                   colourBalancedImage[2184480])
{
  double meanBlue;
  double meanGreen;
  double meanRed;
  double meanTotal;
  double y;
  double *gpu_y;
  unsigned char (*gpu_colourBalancedImage)[2184480];
  unsigned char (*gpu_originalImage)[2184480];
  hipMalloc(&gpu_colourBalancedImage, 2184480UL);
  hipMalloc(&gpu_y, 8UL);
  hipMalloc(&gpu_originalImage, 2184480UL);

  // Input:
  // originalImage = RGB image 984x740x3
  // Output:
  // colourBalancedImage = RGB image 984x740x3
  // Calculate number of pixales in te image
  // Calculate the mean of every colour and take the mean of every colour added
  // togheter
  hipMemcpy(gpu_originalImage, (void *)&originalImage[0], 2184480UL,
             hipMemcpyHostToDevice);
  colourBalance_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  colourBalance_kernel2<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  hipMemcpy(&y, gpu_y, 8UL, hipMemcpyDeviceToHost);
  meanRed = y / 728160.0;
  colourBalance_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  colourBalance_kernel4<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  hipMemcpy(&y, gpu_y, 8UL, hipMemcpyDeviceToHost);
  meanGreen = y / 728160.0;
  colourBalance_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  colourBalance_kernel6<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, gpu_y);
  hipMemcpy(&y, gpu_y, 8UL, hipMemcpyDeviceToHost);
  meanBlue = y / 728160.0;
  meanTotal = ((meanRed + meanGreen) + meanBlue) / 3.0;

  // Create a new image and use the mean value of the colours to calculate the
  // new colours
  meanRed = meanTotal / meanRed;
  y = meanTotal / meanGreen;
  hipMemcpy(gpu_y, &y, 8UL, hipMemcpyHostToDevice);
  colourBalance_kernel7<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_y,
    *gpu_originalImage, meanRed, *gpu_colourBalancedImage);
  colourBalance_kernel8<<<dim3(1423U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_originalImage, meanTotal / meanBlue, *gpu_colourBalancedImage);
  hipMemcpy(&colourBalancedImage[0], gpu_colourBalancedImage, 2184480UL,
             hipMemcpyDeviceToHost);
  hipFree(*gpu_originalImage);
  hipFree(gpu_y);
  hipFree(*gpu_colourBalancedImage);
}

//
// File trailer for colourBalance.cu
//
// [EOF]
//
